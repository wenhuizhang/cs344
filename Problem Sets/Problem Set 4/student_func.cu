#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */


void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{ 
  //TODO
  //PUT YOUR SORT HERE
  
  int threads_per_block = 512;
    int num_blocks = ceil( float(num_elems) /threads_per_block );
    
    unsigned int h_temp[num_elems];  
    unsigned int *d_map_out_group_0;
    unsigned int *d_map_out_group_1;
    
    unsigned int *d_scan_values;
    unsigned int *d_temp0;
    unsigned int *d_temp1;
    unsigned int *d_sum_results;
    unsigned int *d_sort_addresses;
    
    for( int i = 1; i <= 4; i <<= 1){
        map_kernel<<<num_blocks, threads_per_block>>>(d_outputVals, d_map_out_group_0, numElems, i, 0);
        map_kernel<<<num_blocks, threads_per_block>>>(d_outputVals, d_map_out_group_1, numElems, i, 1);
        
        scan_kernel<<<num_blocks, threads_per_block>>>(d_map_out_group_0, d_scan_values, d_temp, d_sum_results, numElems);
        sum_scan_kernel<<<num_blocks, threads_per_block>>>(d_scan_values, d_sum_results, numElems);

        unsigned int addr;
        addr++;
        printf("Addr: %u \n", addr);
        printf("i: %u \n", i);
        
        scan_kernel<<<num_blocks, threads_per_block>>>(d_map_out_group_1, d_scan_values, d_sum_results, numElems);
        sum_scan_kernel<<<num_blocks, threads_per_block>>>(d_scan_values, d_sum_results, numElems);
        
        hipDeviceSynchronize();
        
        map_add_kernel<<<num_blocks, threads_per_block>>>(d_scan_values, d_sort_addresses, d_map_out_group_1, numElems, addr);
        
        resort_addresses<<<num_blocks, threads_per_block>>>(d_outputVals, d_outputPos, d_sort_addresses, d_temp0, d_temp1, numElems, 0);
        resort_addresses<<<num_blocks, threads_per_block>>>(d_outputVals, d_outputPos, d_sort_addresses, d_temp0, d_temp1, numElems, 1);
        
        
        
        
        
        
}
